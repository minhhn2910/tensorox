
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <hip/hip_fp16.h>
#define TYPE1 float
#define TYPE2 float
#define TYPE3 float
#define TYPE4 float
/*
 * 0.085340 ms
8.5531821794
abs err 2.75740586  rel err 0.47576126

 * */
/***
 * 100k
 * Elapsed time : 0.011452 ms
8.4106061017
abs err 2.61482978  rel err 0.45116127

 * 
 * */ 
 /**
  * 1m
  *  0.085744 ms
5.2551581033
abs err 0.54061822  rel err 0.09327796
  * 
  * */
  
  /**
   * 
   *  	0.011482 ms
		5.4107195504
		abs err 0.38505677  rel err 0.06643748
   * 
   * */
 
double fun_ref( double x){
  int k, n = 5;
  double t1;
  double d1 = 1.0;
  t1 = x;
  for ( k = 1; k <= n; k++ ){
      d1 = 2.0 * d1;
      t1 = t1+ sin(d1 * x)/d1;
    }
    return t1;
}
__global__ void fun_gpu(float x[], float y[], int nthreads, float h){
  //y = fun(x)
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int k, n = 5;
  half2 input = __floats2half2_rn(x[tid],x[tid+1]);
  if (tid < nthreads-1) {
    half2 t1;
    half2 d1 = __float2half2_rn(1.0);
    half2 x_temp = input;
    t1 = input;
    for ( k = 1; k <= n; k++ )
      {
        d1 = __float2half2_rn(2.0) * d1;
        half2 sin_res = h2sin(d1 * x_temp);
        t1 = t1 + sin_res/d1;
      }
      
    //float2 output = __half22float2(t1);
    half2 output = t1;
    y[tid] =  hsqrt(__float2half_rn(h*h) + (reinterpret_cast<half&>(output.y) - reinterpret_cast<half&>(output.x)) * (reinterpret_cast<half&>(output.y) - reinterpret_cast<half&>(output.x)));
  }
}


int main( int argc, char **argv) {
  int i,n = 1000000;
  double h, t1, t2, dppi;
  double s1;
  //cuda def
  hipEvent_t start, stop;
  float elapsedTime;
  float *d_x, *d_y, *h_x, *h_y ;
  size_t size = n*sizeof(float);

  h_x = (float*) malloc(size);
  h_y = (float*) malloc(size);
  hipMalloc(&d_x, size);
  hipMalloc(&d_y, size);


  t1 = -1.0;
  dppi = acos(t1);
  s1 = 0.0;
  t1 = 0.0;
  h = dppi / n;
  for ( i = 1; i <= n; i++){
    h_x[i-1] = i * h;
  }
    /* Copy vectors from host memory to device memory */
  hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

  int threads_per_block = 256;

  int block_count = (n + threads_per_block - 1)/threads_per_block;
  hipEventCreate(&start);
  hipEventRecord(start,0);
  for (int i =0;i < 100; i ++)
    fun_gpu<<<block_count, threads_per_block>>>(d_x, d_y, n, h);

  hipDeviceSynchronize();
  hipEventCreate(&stop);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start,stop);
  printf("Elapsed time : %f ms\n" ,elapsedTime/100.0);
  hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);
  s1 = 0.000096; //the first loop
  for ( i = 1; i <= n; i++)
    {
     // t2 = h_y[i-1];
      s1 = s1 + h_y[i-1];// sqrt(h*h + (t2 - t1) * (t2 - t1));
    //  t1 = t2;
    }
  double ref_value = 5.7957763224;
  printf("%.10f\n",s1);
  printf("abs err %.8f  rel err %.8f\n", fabs(s1-ref_value), fabs((s1-ref_value)/ref_value) );
  return 0;
}
