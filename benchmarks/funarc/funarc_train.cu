
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#define TYPE1 double
#define TYPE2 double
#define TYPE3 double
#define TYPE4 double
double fun_ref( double x){
  int k, n = 5;
  double t1;
  double d1 = 1.0;
  t1 = x;
  for ( k = 1; k <= n; k++ ){
      d1 = 2.0 * d1;
      t1 = t1+ sin(d1 * x)/d1;
    }
    return t1;
}
__global__ void fun_gpu(double x[], double y[], int nthreads, double h){
  //y = fun(x)
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int k, n = 5;
  if (tid < nthreads-1) {
    double t1;
    double d1 = 1.0;
    double x_temp = x[tid];
    t1 = x[tid];
    for ( k = 1; k <= n; k++ )
      {
        d1 = 2.0 * d1;
        TYPE4 sin_res = sin(d1 * x_temp);
        t1 = t1 + sin_res/d1;
      }
      
    double t2;
    double d2 = 1.0;
    double x_temp2 = x[tid+1];
    t2 = x[tid+1];
    for ( k = 1; k <= n; k++ )
      {
        d2 = 2.0 * d2;
        TYPE4 sin_res = sin(d2 * x_temp2);
        t2 = t2 + sin_res/d2;
      }
      
    y[tid] =  sqrt(h*h + (t2 - t1) * (t2 - t1));
  }
}


int main( int argc, char **argv) {
  int i,n = 1000000;
  double h, t1, t2, dppi;
  double s1;
  //cuda def
  hipEvent_t start, stop;
  float elapsedTime;
  double *d_x, *d_y, *h_x, *h_y ;
  size_t size = n*sizeof(double);

  h_x = (double*) malloc(size);
  h_y = (double*) malloc(size);
  hipMalloc(&d_x, size);
  hipMalloc(&d_y, size);


  t1 = -1.0;
  dppi = acos(t1);
  s1 = 0.0;
  t1 = 0.0;
  h = dppi / n;
  for ( i = 1; i <= n; i++){
    h_x[i-1] = i * h;
  }
    /* Copy vectors from host memory to device memory */
  hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

  int threads_per_block = 256;

  int block_count = (n + threads_per_block - 1)/threads_per_block;
  hipEventCreate(&start);
  hipEventRecord(start,0);
  for (int i =0;i < 10; i ++)
    fun_gpu<<<block_count, threads_per_block>>>(d_x, d_y, n, h);

  hipDeviceSynchronize();
  hipEventCreate(&stop);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start,stop);
  printf("Elapsed time : %f ms\n" ,elapsedTime);
  hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);
  s1 = 0.000096; //the first loop
  for ( i = 1; i <= n; i++)
    {
     // t2 = h_y[i-1];
      s1 = s1 + h_y[i-1];// sqrt(h*h + (t2 - t1) * (t2 - t1));
    //  t1 = t2;
    }
  double ref_value = 5.7957763224;
  
#define TRAIN_DATA
#ifdef TRAIN_DATA
{
  FILE *fp;
  fp = fopen("input.txt", "w");
  fprintf(fp, "%.8e,%.8e", h_x[0],h_x[1]);
  for (int i=1; i < n-1; i++ ){
	fprintf(fp, ",%.8e,%.8e", h_x[i], h_x[i+1]);   
  }
  fclose(fp);
}

{
  FILE *fp;
  fp = fopen("output.txt", "w");
  fprintf(fp, "%.8e", h_y[0]);
  for (int i=1; i <n-1; i++ ){
    fprintf(fp, ",%.8e", h_y[i]);
  }
  fclose(fp);
}
#endif 

  printf("%.10f\n",s1);
  printf("abs err %.8f  rel err %.8f\n", fabs(s1-ref_value), fabs((s1-ref_value)/ref_value) );
  return 0;
}
