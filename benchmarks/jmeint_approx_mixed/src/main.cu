// Designed by: Amir Yazdanbakhsh
// Date: March 26th - 2015
// Alternative Computing Technologies Lab.
// Georgia Institute of Technology


#include "stdlib.h"
#include <fstream>
#include <iostream>
#include <cstddef>

// Cuda Libraries
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define EPSILON 1e-12 // EPSILON represents the error buffer used to denote a hit

using namespace std;


#include <mma.h>
#include <hip/hip_fp16.h>
using namespace nvcuda;
const int WMMA_M = 32;
const int WMMA_N = 8;
const int WMMA_K = 16;
#define WARP_SIZE 32
#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)


__device__ bool newComputeIntervals(float vv0, float vv1, float vv2, float d0, float d1, float d2, float d0d1, float d0d2, float abc[3], float x0x1[2])
{
	if (d0d1 > 0.0f) {
		// d0d2 <= 0 --> i.e. d0, d1 are on the same side, d2 on the other or on the plane
		abc[0] = vv2;
		abc[1] = (vv0 - vv2) * d2;
		abc[2] = (vv1 - vv2) * d2;
		x0x1[0] = d2 - d0;
		x0x1[1] = d2 - d1;
	} else if (d0d2 > 0.0f) {
		// d0d1 <= 0
		abc[0] = vv1;
		abc[1] = (vv0 - vv1) * d1;
		abc[2] = (vv2 - vv1) * d1;
		x0x1[0] = d1 - d0;
		x0x1[1] = d1 - d2;
	} else if (d1 * d2 > 0.0f || d0 != 0.0f) {
		// d0d1 <= 0 or d0 != 0
		abc[0] = vv0;
		abc[1] = (vv1 - vv0) * d0;
		abc[2] = (vv2 - vv0) * d0;
		x0x1[0] = d0 - d1;
		x0x1[1] = d0 - d2;
	} else if (d1 != 0.0f) {
		abc[0] = vv1;
		abc[1] = (vv0 - vv1) * d1;
		abc[2] = (vv2 - vv1) * d1;
		x0x1[0] = d1 - d0;
		x0x1[1] = d1 - d2;
	} else if (d2 != 0.0f) {
		abc[0] = vv2;
		abc[1] = (vv0 - vv2) * d2;
		abc[2] = (vv1 - vv2) * d2;
		x0x1[0] = d2 - d0;
		x0x1[1] = d2 - d1;
	} else {
		// Triangles are coplanar
		return true;
	}

	return false;
}
__device__ bool edgeEdgeTest(float v0[3], float u0[3], float u1[3], int i0, int i1, float Ax, float Ay)
{
	float Bx = u0[i0] - u1[i0];
	float By = u0[i1] - u1[i1];
	float Cx = v0[i0] - u0[i0];
	float Cy = v0[i1] - u0[i1];
	float f = Ay * Bx - Ax * By;
	float d = By * Cx - Bx * Cy;

	if ((f > 0 && d >= 0 && d <= f) || (f < 0 && d <= 0 && d >= f)) {
		float e = Ax * Cy - Ay * Cx;
		if (f > 0) {
			if (e >= 0 && e <= f)
				return true;
		} else {
			if (e <= 0 && e >= f)
				return true;
		}
	}

	return false;
}

__device__ bool pointInTri(float V0[3], float U0[3], float U1[3], float U2[3], int i0, int i1)
{
	// Check if V0 is inside triangle (U0,U1,U2)

	float a, b, c, d0, d1, d2;
	a = U1[i1] - U0[i1];
	b = -(U1[i0] - U0[i0]);
	c = -a * U0[i0] - b * U0[i1];
	d0 = a * V0[i0] + b * V0[i1] + c;

	a = U2[i1] - U1[i1];
	b = -(U2[i0] - U1[i0]);
	c = -a * U1[i0] - b * U1[i1];
	d1 = a * V0[i0] + b * V0[i1] + c;

	a = U0[i1] - U2[i1];
	b = -(U0[i0] - U2[i0]);
	c = -a * U2[i0] - b * U2[i1];
	d2 = a * V0[i0] + b * V0[i1] + c;

	if ((d0 * d1) > 0.0 && (d0 * d2) > 0.0)
		return true;

	return false;
}

__device__ bool coplanarTriTri(float n[3], float v0[3], float v1[3], float v2[3], float u0[3], float u1[3], float u2[3])
{
	float a[3];
	short i0, i1;
	a[0] = abs(n[0]);
	a[1] = abs(n[1]);
	a[2] = abs(n[2]);

	if (a[0] > a[1]) {
		if (a[0] > a[2]) {
			i0 = 1;
			i1 = 2;
		} else {
			i0 = 0;
			i1 = 1;
		}
	} else {
		if (a[2] > a[1]) {
			i0 = 0;
			i1 = 1;
		} else {
			i0 = 0;
			i1 = 2;
		}
	}

	// Test all edges of triangle 1 against edges of triangle 2
	float aX = v1[i0] - v0[i0];
	float aY = v1[i1] - v0[i1];
	float bX = v2[i0] - v1[i0];
	float bY = v2[i1] - v1[i1];
	float cX = v0[i0] - v2[i0];
	float cY = v0[i1] - v2[i1];
	if ( edgeEdgeTest(v0, u0, u1, i0, i1, aX, aY) || edgeEdgeTest(v0, u1, u2, i0, i1, aX, aY) || edgeEdgeTest(v0, u2, u0, i0, i1, aX, aY) ||
		 edgeEdgeTest(v1, u0, u1, i0, i1, bX, bY) || edgeEdgeTest(v1, u1, u2, i0, i1, bX, bY) || edgeEdgeTest(v1, u2, u0, i0, i1, bX, bY) ||
		 edgeEdgeTest(v2, u0, u1, i0, i1, cX, cY) || edgeEdgeTest(v2, u1, u2, i0, i1, cX, cY) || edgeEdgeTest(v2, u2, u0, i0, i1, cX, cY) )
		return true;

	// Finally, test if either triangle is totally contained in the other
	if (pointInTri(v0, u0, u1, u2, i0, i1) || pointInTri(u0, v0, v1, v2, i0, i1))
		return true;
	return false;

}

__device__ bool jmeint_kernel_impl(float v0[3], float v1[3], float v2[3], float u0[3], float u1[3], float u2[3])
{

    	float e1[3], e2[3], n1[3], n2[3], d[3];
    	float d1, d2;
    	float du0, du1, du2, dv0, dv1, dv2;
    	float du0du1, du0du2, dv0dv1, dv0dv2;

    	float isect1[2];
		float isect2[2];
		short index;
		float vp0, vp1, vp2;
		float up0, up1, up2;
		float bb, cc, max;
		float xx, yy, xxyy, tmp;

		// Compute plane equation of triangle (v0,v1,v2)
		e1[0] = v1[0] - v0[0];
		e1[1] = v1[1] - v0[1];
		e1[2] = v1[2] - v0[2];

		e2[0] = v2[0] - v0[0];
		e2[1] = v2[1] - v0[1];
		e2[2] = v2[2] - v0[2];

		// Cross product: n1 = e1 x e2
		n1[0] = (e1[1] * e2[2]) - (e1[2] * e2[1]);
		n1[1] = (e1[2] * e2[0]) - (e1[0] * e2[2]);
		n1[2] = (e1[0] * e2[1]) - (e1[1] * e2[0]);

		// Plane equation 1: n1.X + d1 = 0
		d1 = -(n1[0] * v0[0] + n1[1] * v0[1] + n1[2] * v0[2]);

		// Put u0,u1,u2 into plane equation 1 to compute signed distances to the plane
		du0 = (n1[0] * u0[0] + n1[1] * u0[1] + n1[2] * u0[2]) + d1;
		du1 = (n1[0] * u1[0] + n1[1] * u1[1] + n1[2] * u1[2]) + d1;
		du2 = (n1[0] * u2[0] + n1[1] * u2[1] + n1[2] * u2[2]) + d1;

		// Coplanarity robustness check
		if ((du0 > 0 && du0 < EPSILON) || (du0 < 0 && du0 > EPSILON))
			du0 = 0.0f;
		if ((du1 > 0 && du1 < EPSILON) || (du1 < 0 && du1 > EPSILON))
			du1 = 0.0f;
		if ((du2 > 0 && du2 < EPSILON) || (du2 < 0 && du2 > EPSILON))
			du2 = 0.0f;

		du0du1 = du0 * du1;
		du0du2 = du0 * du2;

		if (du0du1 > 0.0f && du0du2 > 0.0f) {
			// All 3 have same sign and their values are not equal to 0 --> no intersection
			return false;
		}

		// Compute plane equation of triangle (u0,u1,u2)
		e1[0] = u1[0] - u0[0];
		e1[1] = u1[1] - u0[1];
		e1[2] = u1[2] - u0[2];

		e2[0] = u2[0] - u0[0];
		e2[1] = u2[1] - u0[1];
		e2[2] = u2[2] - u0[2];

		// Cross product: n2 = e1 x e2
		n2[0] = (e1[1] * e2[2]) - (e1[2] * e2[1]);
		n2[1] = (e1[2] * e2[0]) - (e1[0] * e2[2]);
		n2[2] = (e1[0] * e2[1]) - (e1[1] * e2[0]);

		// Plane equation 2: n2.X + d2 = 0
		d2 = -(n2[0] * u0[0] + n2[1] * u0[1] + n2[2] * u0[2]);

		// Put v0,v1,v2 into plane equation 2 to compute signed distances to the plane
		dv0 = (n2[0] * v0[0] + n2[1] * v0[1] + n2[2] * v0[2]) + d2;
		dv1 = (n2[0] * v1[0] + n2[1] * v1[1] + n2[2] * v1[2]) + d2;
		dv2 = (n2[0] * v2[0] + n2[1] * v2[1] + n2[2] * v2[2]) + d2;

		// Coplanarity robustness check
		if ((dv0 > 0 && dv0 < EPSILON) || (dv0 < 0 && dv0 > EPSILON))
			dv0 = 0.0f;
		if ((dv1 > 0 && dv1 < EPSILON) || (dv1 < 0 && dv1 > EPSILON))
			dv1 = 0.0f;
		if ((dv2 > 0 && dv2 < EPSILON) || (dv2 < 0 && dv2 > EPSILON))
			dv2 = 0.0f;

		dv0dv1 = dv0 * dv1;
		dv0dv2 = dv0 * dv2;

		if (dv0dv1 > 0.0f && dv0dv2 > 0.0f) {
			// All 3 have same sign and their values are not equal to 0 --> no intersection
			return false;
		}
		// Compute direction of intersection line --> cross product: d = n1 x n2
		d[0] = (n1[1] * n2[2]) - (n1[2] * n2[1]);
    	d[1] = (n1[2] * n2[0]) - (n1[0] * n2[2]);
    	d[2] = (n1[0] * n2[1]) - (n1[1] * n2[0]);

		// Compute and index to the largest component of d
		index = 0;
		max = abs(d[0]);
		bb = abs(d[1]);
		cc = abs(d[2]);
		if (bb > max) {
			max = bb;
			index = 1;
		}
		if (cc > max) {
			max = cc;
			vp0 = v0[2];
			vp1 = v1[2];
			vp2 = v2[2];
			up0 = u0[2];
			up1 = u1[2];
			up2 = u2[2];
		} else if (index == 1) {
			vp0 = v0[1];
			vp1 = v1[1];
			vp2 = v2[1];
			up0 = u0[1];
			up1 = u1[1];
			up2 = u2[1];
		} else {
			vp0 = v0[0];
			vp1 = v1[0];
			vp2 = v2[0];
			up0 = u0[0];
			up1 = u1[0];
			up2 = u2[0];
		}

		// Compute interval for triangle 1
		float abc[3];
		float x0x1[2];
		if (newComputeIntervals(vp0, vp1, vp2, dv0, dv1, dv2, dv0dv1, dv0dv2, abc, x0x1)) {
			return coplanarTriTri(n1, v0, v1, v2, u0, u1, u2);
		}

		// Compute interval for triangle 2
		float def[3];
		float y0y1[2];
		if (newComputeIntervals(up0, up1, up2, du0, du1, du2, du0du1, du0du2, def, y0y1)) {
			return coplanarTriTri(n1, v0, v1, v2, u0, u1, u2);
		}
		xx = x0x1[0] * x0x1[1];
		yy = y0y1[0] * y0y1[1];
		xxyy = xx * yy;

		tmp = abc[0] * xxyy;
		isect1[0] = tmp + abc[1] * x0x1[1] * yy;
		isect1[1] = tmp + abc[2] * x0x1[0] * yy;

		tmp = def[0] * xxyy;
		isect2[0] = tmp + def[1] * xx * y0y1[1];
		isect2[1] = tmp + def[2] * xx * y0y1[0];

		// Sort isect1 and isect2
		if (isect1[0] > isect1[1]) {
			float f = isect1[0];
			isect1[0] = isect1[1];
			isect1[1] = f;
		}
		if (isect2[0] > isect2[1]) {
			float f = isect2[0];
			isect2[0] = isect2[1];
			isect2[1] = f;
		}

		if (isect1[1] < isect2[0] || isect2[1] < isect1[0])
		{
			return false;
		}
		return true;
}


// The only dimensions currently supported by WMMA

//__constant__ half weight_1_half_d[2][256]; //[32x16]
__constant__ half weight_1_half_d[144];
__constant__ half weight_2_half_d[8];


__constant__ half bias_1_half_d[8];
//__constant__ half bias_3_half_d[1]; only 1 value, no need
#define BIAS_2 0.0617521

__device__ __inline__ half relu( half x){
  return (x>__float2half_rn(0.0))? x:__float2half_rn(0.0) ;
}

__device__ __inline__ half relu_last( half x){
  return (x>=__float2half_rn(0.5))? __float2half_rn(1.0) :__float2half_rn(0.0) ;
}

#define CONST_SCALE __float2half_rn(0.033)

__global__ void jmeint_kernel(float *v0_d, float *v1_d, float *v2_d, float *u0_d, float*u1_d, float*u2_d,
  half *v0_d_half, half *v1_d_half, half *v2_d_half, half *u0_d_half, half *u1_d_half, half *u2_d_half
  , bool* intersect_d, int size, int speed)
{

  if (blockIdx.x %10 < speed){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = tid;
    int real_tid =  threadIdx.x;
     int warp_id = real_tid /32;
     int warp_lane = real_tid %32;

     __shared__ half A[2][8][512];
     __shared__ half weight_1_shared[2][128];
     __shared__ half bias_1_shared[256];
  //   __shared__ half neuron_out[8][512];

     weight_1_shared[0][threadIdx.x] = weight_1_half_d[threadIdx.x];//__float2half_rn(0.0);
   	if(threadIdx.x < 16)
   			weight_1_shared[1][threadIdx.x] = weight_1_half_d[128+threadIdx.x];


     //simple trick no need if/else, read as col_major later
 	   bias_1_shared[real_tid] = bias_1_half_d[warp_id];

     for (int i = 0; i<8 ; i++){
       A[1][i][real_tid] = 0.0;
       A[1][i][real_tid+256] = 0.0;
     }
     __syncthreads();

 	//	wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
 		wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag_col;
 		wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
 	//	wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> acc_frag;
 		wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;

 	//	if (tid < nthreads-1){
 			  A[0][warp_id][warp_lane+ 0*32] = v0_d_half[idx * 3 + 0]*CONST_SCALE;
 			  A[0][warp_id][warp_lane+ 1*32] = v0_d_half[idx * 3 + 1]*CONST_SCALE;
        A[0][warp_id][warp_lane+ 2*32] = v0_d_half[idx * 3 + 2]*CONST_SCALE;

        A[0][warp_id][warp_lane+ 3*32] = v1_d_half[idx * 3 + 0]*CONST_SCALE;
        A[0][warp_id][warp_lane+ 4*32] = v1_d_half[idx * 3 + 1]*CONST_SCALE;
        A[0][warp_id][warp_lane+ 5*32] = v1_d_half[idx * 3 + 2]*CONST_SCALE;

        A[0][warp_id][warp_lane+ 6*32] = v2_d_half[idx * 3 + 0]*CONST_SCALE;
        A[0][warp_id][warp_lane+ 7*32] = v2_d_half[idx * 3 + 1]*CONST_SCALE;
        A[0][warp_id][warp_lane+ 8*32] = v2_d_half[idx * 3 + 2]*CONST_SCALE;

        A[0][warp_id][warp_lane+ 9*32] = u0_d_half[idx * 3 + 0]*CONST_SCALE;
        A[0][warp_id][warp_lane+ 10*32] = u0_d_half[idx * 3 + 1]*CONST_SCALE;
        A[0][warp_id][warp_lane+ 11*32] = u0_d_half[idx * 3 + 2]*CONST_SCALE;

        A[0][warp_id][warp_lane+ 12*32] = u1_d_half[idx * 3 + 0]*CONST_SCALE;
        A[0][warp_id][warp_lane+ 13*32] = u1_d_half[idx * 3 + 1]*CONST_SCALE;
        A[0][warp_id][warp_lane+ 14*32] = u1_d_half[idx * 3 + 2]*CONST_SCALE;

        A[0][warp_id][warp_lane+ 15*32] = u2_d_half[idx * 3 + 0]*CONST_SCALE;
        //next tensor
        A[1][warp_id][warp_lane+ 0*32] = u2_d_half[idx * 3 + 1]*CONST_SCALE;
        A[1][warp_id][warp_lane+ 1*32] = u2_d_half[idx * 3 + 2]*CONST_SCALE;
        __syncthreads();
 		// }
   		wmma::load_matrix_sync(a_frag_col, (const __half*)A[0][warp_id], 32);
   		wmma::load_matrix_sync(b_frag, (const __half*)&weight_1_shared[0], 8);
   		wmma::load_matrix_sync(c_frag, (const half*)&bias_1_shared, 32, wmma::mem_col_major);

   		wmma::mma_sync(c_frag, a_frag_col, b_frag, c_frag);

  		//wmma::store_matrix_sync((half*)neuron_out[warp_id], c_frag, 32,wmma::mem_col_major);
      wmma::store_matrix_sync((half*)A[0][warp_id], c_frag, 32,wmma::mem_col_major);


      wmma::load_matrix_sync(a_frag_col, (const __half*)A[1][warp_id], 32);
      wmma::load_matrix_sync(b_frag, (const __half*)&weight_1_shared[1], 8);

      wmma::load_matrix_sync(c_frag, (const half*)&A[0][warp_id], 32, wmma::mem_col_major);
      wmma::mma_sync(c_frag, a_frag_col, b_frag, c_frag);

      for (int i = 0; i< c_frag.num_elements; i ++)
   			c_frag.x[i] = relu(c_frag.x[i]);

//debug
     wmma::store_matrix_sync((half*)A[0][warp_id], c_frag, 32,wmma::mem_col_major);

      __syncwarp();
      half output1= __float2half_rn(0.0);
      for (int i =0 ; i <8 ; i++ )
        output1 += A[0][warp_id][warp_lane+i*32]*weight_2_half_d[i];
      output1 += __float2half_rn(BIAS_2);
      __syncwarp();

    if (output1 >= __float2half_rn(0.5))
      intersect_d[tid] = true;
    else
      intersect_d[tid] = false;
    //  debug[threadIdx.x] = output1;//neuron_out[0][threadIdx.x];

  }else {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = tid;
    //doing float comp
        float v0[3];
    	float v1[3];
    	float v2[3];
    	float u0[3];
    	float u1[3];
    	float u2[3];

      v0[0] = v0_d[idx * 3 + 0];
      v0[1] =	v0_d[idx * 3 + 1];
      v0[2] = v0_d[idx * 3 + 2];

      v1[0] =	v1_d[idx * 3 + 0];
      v1[1] =	v1_d[idx * 3 + 1];
      v1[2] =	v1_d[idx * 3 + 2];

      v2[0] = v2_d[idx * 3 + 0];
      v2[1] = v2_d[idx * 3 + 1];
      v2[2] =	v2_d[idx * 3 + 2];

      u0[0] = u0_d[idx * 3 + 0];
      u0[1] =	u0_d[idx * 3 + 1];
      u0[2] = u0_d[idx * 3 + 2];

      u1[0] = u1_d[idx * 3 + 0];
      u1[1] = u1_d[idx * 3 + 1];
      u1[2] = u1_d[idx * 3 + 2];

      u2[0] = u2_d[idx * 3 + 0];
      u2[1] = u2_d[idx * 3 + 1];
      u2[2] = u2_d[idx * 3 + 2];

      float parrotInput[18];
        float parrotOutput[1];

        parrotInput[0 ] = v0[0];
        parrotInput[1 ] = v0[1];
        parrotInput[2 ] = v0[2];

        parrotInput[3 ] = v1[0];
        parrotInput[4 ] = v1[1];
        parrotInput[5 ] = v1[2];

        parrotInput[6 ] = v2[0];
        parrotInput[7 ] = v2[1];
        parrotInput[8 ] = v2[2];

        parrotInput[9 ] = u0[0];
        parrotInput[10] = u0[1];
        parrotInput[11] = u0[2];

        parrotInput[12] = u1[0];
        parrotInput[13] = u1[1];
        parrotInput[14] = u1[2];

        parrotInput[15] = u2[0];
        parrotInput[16] = u2[1];
        parrotInput[17] = u2[2];

    #pragma parrot(input, "jmeint_kernel", [18]<-1.0; 1.0>parrotInput)

        intersect_d[idx] = jmeint_kernel_impl(v0, v1, v2, u0, u1, u2);

        if(intersect_d[idx])
        {
          parrotOutput[0] = -0.9;
        }
        else
        {
          parrotOutput[0] = 0.9;
        }

    #pragma parrot(output, "jmeint_kernel", [1]<-0.9; 0.9>parrotOutput)


      if(parrotOutput[0] > 0.0)
      {
        intersect_d[idx] = true;
        return;
      }
      else
      {
        intersect_d[idx] = false;
        return;
      }




  }

}

float weight_1[144] = {0.511078, 0.523341, 0.066438, -0.491299, 0.443703, 0.942849, 0.596723, -0.319132, 0.449167, 0.255444, 0.473568, -0.39083, -0.371555, 0.0453213, 0.0861016, 0.280701, -0.421177, -0.14251, 0.0308778, -0.429457, 0.400507, 0.14804, -0.00634764, 0.356105, 0.416433, -0.136496, -0.0936358, -0.356244, 0.388306, -0.245637, -0.211574, -0.352305, 0.398325, -0.818396, 0.149469, -0.497483, -0.405287, 0.423554, -0.269324, 0.476809, -0.411678, -0.121283, -0.000869494, -0.47774, 0.4155, 0.33816, 0.0341384, 0.387186, 0.461407, 0.335607, -0.0176836, -0.307993, 0.407101, -0.202389, -0.0277943, -0.407035, 0.471145, 0.00753348, 0.449523, -0.432606, -0.372076, 0.273333, -0.0572681, 0.367748, -0.407902, -0.148693, 0.00476071, -0.469246, 0.410727, 0.282667, 0.0214304, 0.385739, -0.481306, -0.251702, -0.082988, 0.387699, -0.434762, -0.138973, -0.103004, 0.380288, -0.453013, 0.219075, 0.360367, 0.445675, 0.389814, -0.227433, -0.0223918, -0.395539, 0.419846, 0.187924, 0.0673863, 0.460819, -0.401764, -0.251448, -0.0710916, -0.353779, -0.479561, -0.214291, -0.0929197, 0.372163, -0.421492, -0.063086, -0.0723261, 0.397593, -0.446763, 0.297718, 0.415162, 0.447785, 0.382669, -0.264714, 0.0129754, -0.404251, 0.413612, 0.14786, 0.0472674, 0.471104, -0.395543, -0.287225, -0.0680886, -0.359023, -0.473956, -0.220853, -0.0993806, 0.410354, -0.403897, -0.14258, -0.0901898, 0.379739, -0.450525, 0.260478, 0.399954, 0.44896, 0.400013, -0.243192, 0.0139935, -0.401857, 0.42331, 0.191439, 0.0510963, 0.471557, -0.409442, -0.21915, -0.0929248, -0.349474};
float bias_1[8] = {0.00521765, -0.0327343, -0.125869, -0.00520308, -0.019705, -0.0615173, 1.24405, -0.0105536};
float weight_2[8] = {0.207281, -0.0984865, 0.0587332, 0.220695, 0.225964, -0.103368, 0.101314, 0.259321};
half weight_1_half[144],bias_1_half[8], weight_2_half[8];

void prepare_weights(){
  for (int i=0;i < 144; i++)
    weight_1_half[i] = __float2half_rn(weight_1[i]);
  for (int i=0;i < 8; i++){
    bias_1_half[i] = __float2half_rn(bias_1[i]);
    weight_2_half[i] = __float2half_rn(weight_2[i]);
  }

}

int main(int argc, char* argv[])
{
	if(argc < 3)
	{
		std::cerr << "Usage: ./jmeint.out <input file locations> <output file>" << std::endl;
		exit(EXIT_FAILURE);
	}

  int speed = 10;
  if (argc >3 )
    speed = atoi(argv[3]);
  printf("running at speed %d \n", speed);
  prepare_weights();

	float (*v0)[3];
	float (*v1)[3];
	float (*v2)[3];
	float (*u0)[3];
	float (*u1)[3];
	float (*u2)[3];

  half (*v0_half)[3];
	half (*v1_half)[3];
	half (*v2_half)[3];
	half (*u0_half)[3];
	half (*u1_half)[3];
	half (*u2_half)[3];

	bool  *intersect;
//  float  *intersect; //debug

	hipError_t cudaStatus;

	int data_size = 0;

	// process the files
	ifstream locations_in_file (argv[1]);
	ofstream intersect_out_file (argv[2]);


	if(locations_in_file.is_open())
	{
		locations_in_file >> data_size;
		std::cout << "# Data Size = " << data_size << std::endl;
	}

//  intersect = new (nothrow) float[data_size]; //debug
	intersect = new (nothrow) bool[data_size];

	// allocate the memory
	v0 = new (nothrow) float[data_size][3];

  v0_half = new (nothrow) half[data_size][3];
	if(v0 == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	// allocate the memory
	v1 = new (nothrow) float[data_size][3];
  v1_half = new (nothrow) half[data_size][3];
	if(v1 == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	// allocate the memory
	v2 = new (nothrow) float[data_size][3];
  v2_half = new (nothrow) half[data_size][3];
	if(v2 == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	// allocate the memory
	u0 = new (nothrow) float[data_size][3];
  u0_half = new (nothrow) half[data_size][3];
	if(u0 == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	// allocate the memory
	u1 = new (nothrow) float[data_size][3];
  u1_half = new (nothrow) half[data_size][3];
	if(u1 == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}
	// allocate the memory
	u2 = new (nothrow) float[data_size][3];
  u2_half = new (nothrow) half[data_size][3];
	if(u2 == NULL)
	{
		std::cerr << "Memory allocation fails!!!" << std::endl;
		exit(EXIT_FAILURE);
	}



	// Prepare
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// add data to the arrays
	int loc_index = 0;
	while(loc_index < data_size)
	{
		locations_in_file 	>> v0[loc_index][0] >> v0[loc_index][1] >> v0[loc_index][2]
							>> v1[loc_index][0] >> v1[loc_index][1] >> v1[loc_index][2]
							>> v2[loc_index][0] >> v2[loc_index][1] >> v2[loc_index][2]
							>> u0[loc_index][0] >> u0[loc_index][1] >> u0[loc_index][2]
							>> u1[loc_index][0] >> u1[loc_index][1] >> u1[loc_index][2]
							>> u2[loc_index][0] >> u2[loc_index][1] >> u2[loc_index][2];

    v0_half[loc_index][0] = __float2half_rn(v0[loc_index][0]);
    v0_half[loc_index][1] = __float2half_rn(v0[loc_index][1]);
    v0_half[loc_index][2] = __float2half_rn(v0[loc_index][2]);

    v1_half[loc_index][0] = __float2half_rn(v1[loc_index][0]);
    v1_half[loc_index][1] = __float2half_rn(v1[loc_index][1]);
    v1_half[loc_index][2] = __float2half_rn(v1[loc_index][2]);

    v2_half[loc_index][0] = __float2half_rn(v2[loc_index][0]);
    v2_half[loc_index][1] = __float2half_rn(v2[loc_index][1]);
    v2_half[loc_index][2] = __float2half_rn(v2[loc_index][2]);


    u0_half[loc_index][0] = __float2half_rn(u0[loc_index][0]);
    u0_half[loc_index][1] = __float2half_rn(u0[loc_index][1]);
    u0_half[loc_index][2] = __float2half_rn(u0[loc_index][2]);

    u1_half[loc_index][0] = __float2half_rn(u1[loc_index][0]);
    u1_half[loc_index][1] = __float2half_rn(u1[loc_index][1]);
    u1_half[loc_index][2] = __float2half_rn(u1[loc_index][2]);

    u2_half[loc_index][0] = __float2half_rn(u2[loc_index][0]);
    u2_half[loc_index][1] = __float2half_rn(u2[loc_index][1]);
    u2_half[loc_index][2] = __float2half_rn(u2[loc_index][2]);


		loc_index++;
	}


	std::cout << "# Coordinates are read from file..." << std::endl;

	// memory allocations on the host
	float *v0_d;
	float *v1_d;
	float *v2_d;
	float *u0_d;
	float *u1_d;
	float *u2_d;

  half *v0_d_half;
  half *v1_d_half;
  half *v2_d_half;
  half *u0_d_half;
  half *u1_d_half;
  half *u2_d_half;

	bool  *intersect_d;
//  float  *intersect_d; //debug


	hipMalloc((void**) &v0_d, data_size * 3 * sizeof(float));
	hipMalloc((void**) &v1_d, data_size * 3 * sizeof(float));
	hipMalloc((void**) &v2_d, data_size * 3 * sizeof(float));
	hipMalloc((void**) &u0_d, data_size * 3 * sizeof(float));
	hipMalloc((void**) &u1_d, data_size * 3 * sizeof(float));
	hipMalloc((void**) &u2_d, data_size * 3 * sizeof(float));

  hipMalloc((void**) &v0_d_half, data_size * 3 * sizeof(half));
	hipMalloc((void**) &v1_d_half, data_size * 3 * sizeof(half));
	hipMalloc((void**) &v2_d_half, data_size * 3 * sizeof(half));
	hipMalloc((void**) &u0_d_half, data_size * 3 * sizeof(half));
	hipMalloc((void**) &u1_d_half, data_size * 3 * sizeof(half));
	hipMalloc((void**) &u2_d_half, data_size * 3 * sizeof(half));


//debug
  float* debug_h, *debug_d;
  debug_h = (float*)malloc(data_size * sizeof(float));
  hipMalloc((void**) &debug_d, data_size * sizeof(float));


	hipMalloc((void**) &intersect_d, data_size * sizeof(bool));
//  cudaMalloc((void**) &intersect_d, data_size * sizeof(float));//debug


	std::cout << "# Memory allocation on GPU is done..." << std::endl;

	hipMemcpy(v0_d, v0, data_size * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(v1_d, v1, data_size * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(v2_d, v2, data_size * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(u0_d, u0, data_size * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(u1_d, u1, data_size * 3 * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(u2_d, u2, data_size * 3 * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(v0_d_half, v0_half, data_size * 3 * sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(v1_d_half, v1_half, data_size * 3 * sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(v2_d_half, v2_half, data_size * 3 * sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(u0_d_half, u0_half, data_size * 3 * sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(u1_d_half, u1_half, data_size * 3 * sizeof(half), hipMemcpyHostToDevice);
	hipMemcpy(u2_d_half, u2_half, data_size * 3 * sizeof(half), hipMemcpyHostToDevice);



  hipMemcpyToSymbol(HIP_SYMBOL(weight_1_half_d), &weight_1_half, 144 * sizeof(half));
  hipMemcpyToSymbol(HIP_SYMBOL(weight_2_half_d), &weight_2_half, 8 * sizeof(half));
  hipMemcpyToSymbol(HIP_SYMBOL(bias_1_half_d), &bias_1_half, 8 * sizeof(half));



	std::cout << "# Data are transfered to GPU..." << std::endl;

	dim3 dimBlock	( 256, 1 );
	dim3 dimGrid	( data_size / 256, 1 );
//	dim3 dimGrid	(1, 1 ); //debug


	hipEventRecord(start, 0);

#pragma parrot.start("jmeint_kernel")

	jmeint_kernel<<<dimGrid, dimBlock>>>(v0_d, v1_d, v2_d, u0_d, u1_d, u2_d,
    v0_d_half, v1_d_half, v2_d_half, u0_d_half, u1_d_half, u2_d_half
     , intersect_d, data_size, speed);

#pragma parrot.end("jmeint_kernel")

	cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
       	std::cout << "Something was wrong! Error code: " << cudaStatus << std::endl;
    }

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << "# Elapsed Time in `jmeint` kernel = " << elapsedTime << std::endl;
	std::cout << "# GPU computation is done ..." << std::endl;

	hipMemcpy(intersect, intersect_d, data_size * sizeof(bool), hipMemcpyDeviceToHost);
//  cudaMemcpy(intersect, intersect_d, data_size * sizeof(float), cudaMemcpyDeviceToHost); //debug




	for(int i = 0; i < data_size; i++)
	{
		intersect_out_file << intersect[i];
		intersect_out_file << std::endl;
	}

	// close files
	locations_in_file.close();
	intersect_out_file.close();

	// de-allocate the memory
	delete[] v0;
	delete[] v1;
	delete[] v2;
	delete[] u0;
	delete[] u1;
	delete[] u2;
	delete[] intersect;

	// de-allocate cuda memory
	hipFree(v0_d);
	hipFree(v1_d);
	hipFree(v2_d);
	hipFree(u0_d);
	hipFree(u1_d);
	hipFree(u2_d);
	hipFree(intersect_d);

	std::cout << "Thank you..." << std::endl;
}
